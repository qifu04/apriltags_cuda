#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <opencv2/opencv.hpp>

#include "apriltag_gpu.h"
#include "apriltag_utils.h"

extern "C" {
#include "apriltag.h"
}

int main(int argc, char **argv) {
  if (argc < 2) {
    std::cerr << "Usage: " << argv[0] << " image_path\n";
    return 1;
  }
  const char *tag_family = "tag36h11";

  cv::Mat bgr = cv::imread(argv[1], cv::IMREAD_COLOR);
  if (bgr.empty()) {
    std::cerr << "Failed to load image: " << argv[1] << "\n";
    return 1;
  }
  int width = bgr.cols;
  int height = bgr.rows;
  std::cout << "Image size: " << width << "x" << height << std::endl;
  if (width % 8 || height % 8) {
    std::cerr << "Image dimensions must be multiples of 8\n";
    return 1;
  }

  cv::Mat gray;
  cv::cvtColor(bgr, gray, cv::COLOR_BGR2GRAY);
  image_u8_t im{gray.cols, gray.rows, gray.cols, gray.data};

  apriltag_family_t *tf = nullptr;
  if (!setup_tag_family(&tf, tag_family)) {
    std::cerr << "Could not setup tag family\n";
    return 1;
  }
  apriltag_detector_t *td = apriltag_detector_create();
  apriltag_detector_add_family(td, tf);
  td->quad_decimate = 1.0;
  td->quad_sigma = 0.0;
  td->nthreads = 1;
  td->debug = false;
  td->refine_edges = true;
  td->wp = workerpool_create(1);

  auto cpu_start = std::chrono::steady_clock::now();
  zarray_t *cpu_detections = apriltag_detector_detect(td, &im);
  auto cpu_end = std::chrono::steady_clock::now();
  std::cout << "CPU detections: " << zarray_size(cpu_detections) << " time "
            << std::chrono::duration_cast<std::chrono::milliseconds>(cpu_end -
                                                                     cpu_start)
                   .count()
            << " ms" << std::endl;

  cv::Mat yuyv;
  cv::cvtColor(bgr, yuyv, cv::COLOR_BGR2YUV_YUYV);

  frc971::apriltag::CameraMatrix cam{};
  cam.fx = width;
  cam.fy = height;
  cam.cx = width / 2.0;
  cam.cy = height / 2.0;
  frc971::apriltag::DistCoeffs dist{};

  frc971::apriltag::GpuDetector gpu(width, height, td, cam, dist);

  auto gpu_start = std::chrono::steady_clock::now();
  gpu.Detect(yuyv.data);
  auto gpu_end = std::chrono::steady_clock::now();
  const zarray_t *gpu_detections = gpu.Detections();
  std::cout << "GPU detections: " << zarray_size(gpu_detections) << " time "
            << std::chrono::duration_cast<std::chrono::milliseconds>(gpu_end -
                                                                     gpu_start)
                   .count()
            << " ms" << std::endl;

  bool match = zarray_size(cpu_detections) == zarray_size(gpu_detections);
  if (match) {
    for (int i = 0; i < zarray_size(cpu_detections); ++i) {
      apriltag_detection_t *cd;
      apriltag_detection_t *gd;
      zarray_get(cpu_detections, i, &cd);
      zarray_get(gpu_detections, i, &gd);
      if (cd->id != gd->id || std::abs(cd->c[0] - gd->c[0]) > 0.5 ||
          std::abs(cd->c[1] - gd->c[1]) > 0.5) {
        match = false;
        break;
      }
    }
  }
  std::cout << "Results " << (match ? "match" : "do not match") << std::endl;

  apriltag_detections_destroy(cpu_detections);
  apriltag_detector_destroy(td);
  teardown_tag_family(&tf, tag_family);
  return match ? 0 : 2;
}
