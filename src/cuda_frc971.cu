#include "cuda_frc971.h"
#include "gflags/gflags.h"
#include "glog/logging.h"

DEFINE_bool(
    sync, false,
    "If true, force synchronization after each step to isolate errors better.");

namespace frc971::apriltag {

size_t overall_memory = 0;

void CheckAndSynchronize(std::string_view message) {
  CHECK_CUDA(hipDeviceSynchronize()) << message;
  CHECK_CUDA(hipGetLastError()) << message;
}

void MaybeCheckAndSynchronize() {
  if (FLAGS_sync) CheckAndSynchronize();
}

void MaybeCheckAndSynchronize(std::string_view message) {
  if (FLAGS_sync) CheckAndSynchronize(message);
}

}  // namespace frc971::apriltag
