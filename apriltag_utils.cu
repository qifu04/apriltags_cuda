#include <iomanip>
#include <iostream>

#include "apriltag_utils.h"

using namespace std;
using namespace cv;

void setup_tag_family(apriltag_family_t **tf, const char *famname) {
  if (!strcmp(famname, "tag36h11")) {
    *tf = tag36h11_create();
  } else if (!strcmp(famname, "tag25h9")) {
    *tf = tag25h9_create();
  } else if (!strcmp(famname, "tag16h5")) {
    *tf = tag16h5_create();
  } else if (!strcmp(famname, "tagCircle21h7")) {
    *tf = tagCircle21h7_create();
  } else if (!strcmp(famname, "tagCircle49h12")) {
    *tf = tagCircle49h12_create();
  } else if (!strcmp(famname, "tagStandard41h12")) {
    *tf = tagStandard41h12_create();
  } else if (!strcmp(famname, "tagStandard52h13")) {
    *tf = tagStandard52h13_create();
  } else if (!strcmp(famname, "tagCustom48h12")) {
    *tf = tagCustom48h12_create();
  } else {
    printf("Unrecognized tag family name. Use e.g. \"tag36h11\".\n");
    exit(-1);
  }
}

void teardown_tag_family(apriltag_family_t **tf, const char *famname) {
  if (!strcmp(famname, "tag36h11")) {
    tag36h11_destroy(*tf);
  } else if (!strcmp(famname, "tag25h9")) {
    tag25h9_destroy(*tf);
  } else if (!strcmp(famname, "tag16h5")) {
    tag16h5_destroy(*tf);
  } else if (!strcmp(famname, "tagCircle21h7")) {
    tagCircle21h7_destroy(*tf);
  } else if (!strcmp(famname, "tagCircle49h12")) {
    tagCircle49h12_destroy(*tf);
  } else if (!strcmp(famname, "tagStandard41h12")) {
    tagStandard41h12_destroy(*tf);
  } else if (!strcmp(famname, "tagStandard52h13")) {
    tagStandard52h13_destroy(*tf);
  } else if (!strcmp(famname, "tagCustom48h12")) {
    tagCustom48h12_destroy(*tf);
  }
}

void draw_detection_outlines(Mat &im, zarray_t *detections) {
  for (int i = 0; i < zarray_size(detections); i++) {
    apriltag_detection_t *det;
    zarray_get(detections, i, &det);
    line(im, Point(det->p[0][0], det->p[0][1]),
         Point(det->p[1][0], det->p[1][1]), Scalar(0, 0xff, 0), 2);
    line(im, Point(det->p[0][0], det->p[0][1]),
         Point(det->p[3][0], det->p[3][1]), Scalar(0, 0, 0xff), 2);
    line(im, Point(det->p[1][0], det->p[1][1]),
         Point(det->p[2][0], det->p[2][1]), Scalar(0xff, 0, 0), 2);
    line(im, Point(det->p[2][0], det->p[2][1]),
         Point(det->p[3][0], det->p[3][1]), Scalar(0xff, 0, 0), 2);

    stringstream ss;
    ss << det->id;
    String text = ss.str();
    int fontface = FONT_HERSHEY_SCRIPT_SIMPLEX;
    double fontscale = 1.0;
    int baseline;
    Size textsize = getTextSize(text, fontface, fontscale, 2, &baseline);
    putText(
        im, text,
        Point(det->c[0] - textsize.width / 2, det->c[1] + textsize.height / 2),
        fontface, fontscale, Scalar(0xff, 0x99, 0), 2);
  }
}

void print_detections(zarray_t *detections) {
  for (int i = 0; i < zarray_size(detections); i++) {
    apriltag_detection_t *det;
    zarray_get(detections, i, &det);
    std::cout << "tag #: " << det->id << std::endl;
    std::cout << "hamming: " << det->hamming << std::endl;
    std::cout << "margin: " << det->decision_margin << std::endl;
    std::cout << "center: " << det->c[0] << "," << det->c[1] << std::endl;
    for (size_t j = 0; j < det->H->ncols; ++j) {
      std::cout << std::endl;
      for (size_t k = 0; k < det->H->nrows; ++k) {
        std::cout << matd_get(det->H, j, k) << " ";
      }
    }
    std::cout << std::endl;
  }
}
